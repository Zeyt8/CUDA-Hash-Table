#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <ctime>
#include <sstream>
#include <string>
#include "test_map.hpp"
#include "gpu_hashtable.hpp"

using namespace std;

/*
Allocate CUDA memory only through glbGpuAllocator
hipMalloc -> glbGpuAllocator->_cudaMalloc
hipMallocManaged -> glbGpuAllocator->_cudaMallocManaged
hipFree -> glbGpuAllocator->_cudaFree
*/

__device__ static unsigned int fnvHash(const char* str)
{
    const size_t length = sizeof(uint32_t);
    unsigned int hash = 2166136261u;
    for (size_t i = 0; i < length; ++i)
    {
        hash ^= *str++;
        hash *= 16777619u;
    }
	hash &= 0x7FFFFFFF;
    return hash;
}

/**
 * Function constructor GpuHashTable
 * Performs init
 * Example on using wrapper allocators _cudaMalloc and _cudaFree
 */
GpuHashTable::GpuHashTable(int size) {
	// alloc table with malloc
	HashTableItem* temp = (HashTableItem*)malloc(size * sizeof(HashTableItem));
	// initialize values
	for (int i = 0; i < size; i++) {
		temp[i].key = 0;
		temp[i].value = 0;
	}
	glbGpuAllocator->_cudaMalloc((void**)&table, size * sizeof(HashTableItem));
	// move temp to gpu
	hipMemcpy(table, temp, size * sizeof(HashTableItem), hipMemcpyHostToDevice);
	free(temp);
	GpuHashTable::size = size;
	count = 0;
}

/**
 * Function desctructor GpuHashTable
 */
GpuHashTable::~GpuHashTable() {
	glbGpuAllocator->_cudaFree(table);
}

/**
 * Function reshape
 * Performs resize of the hashtable based on load factor
 */
__global__ void reshapeKernel(HashTableItem* newTable, HashTableItem* table, int numBucketsReshape)
{
	// calculate index
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	// if index is out of bounds, return
	if (index >= numBucketsReshape) {
		return;
	}
	// if previous entry is empty, return
	if (table[index].key == 0) {
		return;
	}
	// recalculate hash
	unsigned int hash = fnvHash((char*)&table[index].key) % numBucketsReshape;
	// find place to insert
	while (true) {
		uint32_t prev = atomicCAS(&newTable[hash].key, 0, table[index].key);
		if (prev == 0 || prev == table[index].key) {
			newTable[hash].value = table[index].value;
			return;
		}
		hash = (hash + 1) % numBucketsReshape;
	}
}

void GpuHashTable::reshape(int numBucketsReshape) {
	// alloc temp with malloc
	HashTableItem* temp = (HashTableItem*)malloc(numBucketsReshape * sizeof(HashTableItem));
	// initialize values
	for (int i = 0; i < numBucketsReshape; i++) {
		temp[i].key = 0;
		temp[i].value = 0;
	}
	// alloc new table
	HashTableItem* newTable;
	glbGpuAllocator->_cudaMalloc((void**)&newTable, numBucketsReshape * sizeof(HashTableItem));
	// move temp to gpu
	hipMemcpy(newTable, temp, numBucketsReshape * sizeof(HashTableItem), hipMemcpyHostToDevice);
	free(temp);
	// call kernel
	reshapeKernel<<<size / 256 + 1, 256>>>(newTable, table, numBucketsReshape);
	hipDeviceSynchronize();
	// update table
	glbGpuAllocator->_cudaFree(table);
	table = newTable;
	size = numBucketsReshape;
}

/**
 * Function insertBatch
 * Inserts a batch of key:value, using GPU and wrapper allocators
 */
__global__ void insertBatchKernel(HashTableItem* table, int size, int* keys, int* values, int numKeys, int* added)
{
	// calculate index
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	// if index is out of bounds, return
	if (index >= numKeys) {
		return;
	}
	// calculate hash
	int key = keys[index];
	int value = values[index];
	unsigned int hash = fnvHash((char*)&key) % size;
	// find place to insert
	while (true) {
		uint32_t prev = atomicCAS(&table[hash].key, 0, key);
		if (prev == 0) {
			// entry did not exist
			table[hash].value = value;
			atomicAdd(added, 1);
			return;
		}
		else if (prev == key) {
			// entry already exists
			table[hash].value = value;
			return;
		}
		hash = (hash + 1) % size;
	}
}

bool GpuHashTable::insertBatch(int *keys, int* values, int numKeys) {
	// check if we need to reshape
	if (count + numKeys > size) {
		int newSize = size;
		while (count + numKeys > newSize) {
			newSize *= 2;
		}
		reshape(newSize);
	}
	// move keys and values to GPU
	int* keysDevice;
	glbGpuAllocator->_cudaMalloc((void**)&keysDevice, numKeys * sizeof(int));
	hipMemcpy(keysDevice, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);
	int* valuesDevice;
	glbGpuAllocator->_cudaMalloc((void**)&valuesDevice, numKeys * sizeof(int));
	hipMemcpy(valuesDevice, values, numKeys * sizeof(int), hipMemcpyHostToDevice);
	// keep track of how many keys were actually added
	int* added;
	glbGpuAllocator->_cudaMallocManaged((void**)&added, sizeof(int));
	*added = 0;
	// call kernel
	insertBatchKernel<<<numKeys / 256 + 1, 256>>>(table, size, keysDevice, valuesDevice, numKeys, added);
	hipDeviceSynchronize();
	hipError_t err;
	if (0 != (err = hipGetLastError()))
    {
        cerr << "[insertBatch] Error inserting batch" << endl;
        cerr << hipGetErrorString(err) << endl;
    }
	// update count
	count += *added;
	// cleanup
	glbGpuAllocator->_cudaFree(added);
	glbGpuAllocator->_cudaFree(keysDevice);
	glbGpuAllocator->_cudaFree(valuesDevice);
	return true;
}

/**
 * Function getBatch
 * Gets a batch of key:value, using GPU
 */
__global__ void getBatchKernel(HashTableItem* table, int size, int* keys, int* values)
{
	// calculate index
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	// if index is out of bounds, return
	if (index >= size) {
		return;
	}
	// calculate hash
	int key = keys[index];
	int hash = fnvHash((char*)key) % size;
	// find place to insert
	while (true) {
		if (table[hash].key == key) {
			// entry exists
			values[index] = table[hash].value;
			return;
		}
		else if (table[hash].key == 0) {
			// entry does not exist
			values[index] = 0;
			return;
		}
		hash = (hash + 1) % size;
	}
}

int* GpuHashTable::getBatch(int* keys, int numKeys) {
	// move keys to GPU
	int* keysDevice;
	glbGpuAllocator->_cudaMalloc((void**)&keysDevice, numKeys * sizeof(int));
	hipMemcpy(keysDevice, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);
	// alloc return values
	int* values;
	glbGpuAllocator->_cudaMalloc((void**)&values, numKeys * sizeof(int));
	// call kernel
	getBatchKernel<<<numKeys / 256 + 1, 256>>>(table, size, keysDevice, values);
	hipDeviceSynchronize();
	// alloc values on host
	int* valuesHost = (int*)malloc(numKeys * sizeof(int));
	// move values to host
	hipMemcpy(valuesHost, values, numKeys * sizeof(int), hipMemcpyDeviceToHost);
	// cleanup
	glbGpuAllocator->_cudaFree(values);
	glbGpuAllocator->_cudaFree(keysDevice);
	return values;
}

float GpuHashTable::loadFactor() {
	return (float)count / size;
}