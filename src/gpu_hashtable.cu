#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <ctime>
#include <sstream>
#include <string>
#include "test_map.hpp"
#include "gpu_hashtable.hpp"

using namespace std;

/*
Allocate CUDA memory only through glbGpuAllocator
hipMalloc -> glbGpuAllocator->_cudaMalloc
hipMallocManaged -> glbGpuAllocator->_cudaMallocManaged
hipFree -> glbGpuAllocator->_cudaFree
*/

__device__ static int fnvHash(const char* str)
{
    unsigned int hash = 2166136261u;
	hash ^= *(str + 0);
    hash *= 16777619u;
	hash ^= *(str + 1);
    hash *= 16777619u;
	hash ^= *(str + 2);
    hash *= 16777619u;
	hash ^= *(str + 3);
    hash *= 16777619u;
	hash &= 0x7FFFFFFF;
    return (int)hash;
}

/**
 * Function constructor GpuHashTable
 * Performs init
 * Example on using wrapper allocators _cudaMalloc and _cudaFree
 */
GpuHashTable::GpuHashTable(int size) {
	glbGpuAllocator->_cudaMalloc((void**)&table, size * sizeof(HashTableItem));
	hipMemset(table, 0, size * sizeof(HashTableItem));
	capacity = size;
	count = 0;
}

/**
 * Function desctructor GpuHashTable
 */
GpuHashTable::~GpuHashTable() {
	glbGpuAllocator->_cudaFree(table);
}

/**
 * Function reshape
 * Performs resize of the hashtable based on load factor
 */
__global__ void reshapeKernel(HashTableItem* newTable, HashTableItem* table, int size, int numBucketsReshape)
{
	// calculate index
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	// if index is out of bounds, return
	if (index >= size) {
		return;
	}
	// if previous entry is empty, return
	if (table[index].key == 0) {
		return;
	}
	// recalculate hash
	int hash = fnvHash((char*)&table[index].key) % numBucketsReshape;
	// find place to insert
	while (true) {
		int prev = atomicCAS(&newTable[hash].key, 0, table[index].key);
		if (prev == 0) {
			newTable[hash].value = table[index].value;
			return;
		}
		hash = (hash + 1) % numBucketsReshape;
	}
}

void GpuHashTable::reshape(int numBucketsReshape) {
	// alloc new table
	HashTableItem* newTable;
	glbGpuAllocator->_cudaMalloc((void**)&newTable, numBucketsReshape * sizeof(HashTableItem));
	hipMemset(newTable, 0, numBucketsReshape * sizeof(HashTableItem));
	// call kernel
	reshapeKernel<<<(capacity + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(newTable, table, capacity, numBucketsReshape);
	hipDeviceSynchronize();
	// update table
	glbGpuAllocator->_cudaFree(table);
	table = newTable;
	capacity = numBucketsReshape;
}

/**
 * Function insertBatch
 * Inserts a batch of key:value, using GPU and wrapper allocators
 */
__global__ void insertBatchKernel(HashTableItem* table, int size, int* keys, int* values, int numKeys, int* added)
{
	// calculate index
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	// if index is out of bounds, return
	if (index >= numKeys) {
		return;
	}
	// calculate hash
	int key = keys[index];
	int value = values[index];
	int hash = fnvHash((char*)&key) % size;
	// find place to insert
	while (true) {
		int prev = atomicCAS(&table[hash].key, 0, key);
		if (prev == 0) {
			// entry did not exist
			table[hash].value = value;
			atomicAdd(added, 1);
			return;
		}
		else if (prev == key) {
			// entry already exists
			table[hash].value = value;
			return;
		}
		hash = (hash + 1) % size;
	}
}

bool GpuHashTable::insertBatch(int *keys, int* values, int numKeys) {
	// check if we need to reshape
	if (count + numKeys > capacity * LOAD_FACTOR) {
		int newSize = capacity;
		while (count + numKeys > newSize * LOAD_FACTOR) {
			newSize *= 2;
		}
		reshape(newSize);
	}
	// move keys and values to GPU
	int* keysDevice;
	glbGpuAllocator->_cudaMalloc((void**)&keysDevice, numKeys * sizeof(int));
	hipMemcpy(keysDevice, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);
	int* valuesDevice;
	glbGpuAllocator->_cudaMalloc((void**)&valuesDevice, numKeys * sizeof(int));
	hipMemcpy(valuesDevice, values, numKeys * sizeof(int), hipMemcpyHostToDevice);
	// keep track of how many keys were actually added
	int* added;
	glbGpuAllocator->_cudaMalloc((void**)&added, sizeof(int));
	hipMemset(added, 0, sizeof(int));
	// call kernel
	insertBatchKernel<<<(numKeys + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(table, capacity, keysDevice, valuesDevice, numKeys, added);
	hipDeviceSynchronize();
	// update count
	int* addedHost = (int*)malloc(sizeof(int));
	hipMemcpy(addedHost, added, sizeof(int), hipMemcpyDeviceToHost);
	count += *addedHost;
	// cleanup
	free(addedHost);
	glbGpuAllocator->_cudaFree(added);
	glbGpuAllocator->_cudaFree(keysDevice);
	glbGpuAllocator->_cudaFree(valuesDevice);
	return true;
}

/**
 * Function getBatch
 * Gets a batch of key:value, using GPU
 */
__global__ void getBatchKernel(HashTableItem* table, int size, int* keys, int* values, int numKeys)
{
	// calculate index
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	// if index is out of bounds, return
	if (index >= numKeys) {
		return;
	}
	// calculate hash
	int key = keys[index];
	int hash = fnvHash((char*)&key) % size;
	int start = hash;
	// find place to insert
	while (true) {
		if (table[hash].key == key) {
			// entry exists
			values[index] = table[hash].value;
			return;
		}
		else if (table[hash].key == 0) {
			// entry does not exist
			values[index] = 0;
			return;
		}
		hash = (hash + 1) % size;
		if (hash == start) {
			// we have looped through the entire table
			values[index] = 0;
			return;
		}
	}
}

int* GpuHashTable::getBatch(int* keys, int numKeys) {
	// move keys to GPU
	int* keysDevice;
	glbGpuAllocator->_cudaMalloc((void**)&keysDevice, numKeys * sizeof(int));
	hipMemcpy(keysDevice, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);
	// alloc return values
	int* values;
	glbGpuAllocator->_cudaMalloc((void**)&values, numKeys * sizeof(int));
	// call kernel
	getBatchKernel<<<(numKeys + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(table, capacity, keysDevice, values, numKeys);
	hipDeviceSynchronize();
	// alloc values on host
	int* valuesHost = (int*)malloc(numKeys * sizeof(int));
	// move values to host
	hipMemcpy(valuesHost, values, numKeys * sizeof(int), hipMemcpyDeviceToHost);
	// cleanup
	glbGpuAllocator->_cudaFree(values);
	glbGpuAllocator->_cudaFree(keysDevice);
	return valuesHost;
}

float GpuHashTable::loadFactor() {
	return (float)count / capacity;
}